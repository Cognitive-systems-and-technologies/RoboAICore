#include "hip/hip_runtime.h"
#include "Tensor.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <hiprand/hiprand_kernel.h>

#ifdef __NVCC__
Tensor Tensor_FromDataGPU(shape s, const float* data) 
{
	Tensor t = Tensor_CreateGPU(s, 0.f);
	hipMemcpy(t.w, data, sizeof(float)*t.n, hipMemcpyHostToDevice);
	return t;
}

__global__ void Tensor_FillKernel(int limit, float *w, float v) 
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(i<limit)
		w[i] = v;
}

void Tensor_FillGPU(Tensor *v, float c) 
{
	int threadsPerBlockX = 32;

	dim3 gridDim(ceil(v->n / (float)threadsPerBlockX), 1, 1);
	dim3 blockDim(threadsPerBlockX, 1, 1);
	Tensor_FillKernel KERNEL_CALL(gridDim, blockDim) (v->n, v->w, c);
	hipDeviceSynchronize();
}

void Tensor_FillArrayGPU(float* v, int n, float c)
{
	int threadsPerBlockX = 32;
	if (n < threadsPerBlockX) threadsPerBlockX = 1;
	dim3 gridDim(ceil(n / (float)threadsPerBlockX), 1, 1);
	dim3 blockDim(threadsPerBlockX, 1, 1);
	Tensor_FillKernel KERNEL_CALL(gridDim, blockDim) (n, v, c);
	hipDeviceSynchronize();
}

Tensor Tensor_CreateGPU(shape s, float c)
{
	Tensor v;
	v.s.w = s.w;
	v.s.h = s.h;
	v.s.d = s.d;
	v.n = s.w * s.h * s.d;
	v.sumdw = 0;

	v.w = NULL; 
	v.dw = NULL; 
	//v.vt = NULL;
	v.tData = NULL;

	if (hipMalloc((void**)&v.w, v.n * sizeof(float)) != hipSuccess) printf("Tensor weights allocation error\n");
	else Tensor_FillGPU(&v, c);
	if (hipMalloc((void**)&v.dw, v.n * sizeof(float)) != hipSuccess) printf("Tensor grads allocation error\n");
	else hipMemset(v.dw, 0, sizeof(float) * v.n);
	//if (hipMalloc((void**)&v.vt, v.n * sizeof(float)) != hipSuccess) printf("Tensor additions allocation error\n");
	//else hipMemset(v.vt, 0, sizeof(float) * v.n);

	return v;
}
void Tensor_FreeGPU(Tensor* v)
{
	if (hipFree(v->w) != hipSuccess) printf("Tensor weights free error\n");
	else v->w = NULL;
	if (hipFree(v->dw) != hipSuccess) printf("Tensor grads free error\n");
	else v->dw = NULL;
	//if (hipFree(v->vt) != hipSuccess) printf("Tensor additions free error\n");
	//else v->vt = NULL;
}

void Tensor_CopyDataGPU(Tensor* dst, Tensor* src)
{
	hipMemcpy(dst->w, src->w, sizeof(float) * src->n, hipMemcpyDeviceToDevice);
}
#endif

//print weights
#ifdef __NVCC__
__global__ void TPrintKernel(float* w, int n)
{
	printf("[");
	for (int i = 0; i < n; i++)
		printf("%f, ", w[i]);
	printf("]\n");
}
void Tensor_PrintGPU(Tensor* v)
{
	TPrintKernel KERNEL_CALL_ONCE(v->w, v->n);
	hipDeviceSynchronize();
}
void Tensor_PrintArrayGPU(float* v, int n)
{
	TPrintKernel KERNEL_CALL_ONCE(v, n);
	hipDeviceSynchronize();
}
#endif

//random weights
#ifdef __NVCC__
__global__ void xavier_rand_kernel(void* globalState, float* w, int n)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < n) {
		//calculate the range for the weights
		float lower = -(1.0f / sqrtf(n));
		float upper = (1.0f / sqrtf(n));
		hiprandState localState = ((hiprandState*)globalState)[i];
		float num = hiprand_uniform(&localState);
		//scale to the desired range
		float scaled = lower + num * (upper - lower);
		w[i] = scaled;
		((hiprandState*)globalState)[i] = localState;
	}
}

__global__ void setup_rng_kernel(int limit, void* state)
{
	int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (id < limit) {
		hiprandState* ls = (hiprandState*)state;
		hiprand_init(clock(), id, 0, &ls[id]);
	}
}
void Tensor_Xavier_RandGPU(float *w, int n)
{
	hiprandState* devStates;
	hipMalloc(&devStates, n * sizeof(hiprandState));
	setup_rng_kernel KERNEL_CALL(n, 1) (n, devStates);
	hipDeviceSynchronize();

	int threadsPerBlockX = 32;

	dim3 gridDim(ceil(n / (float)threadsPerBlockX), 1, 1);
	dim3 blockDim(threadsPerBlockX, 1, 1);
	xavier_rand_kernel KERNEL_CALL(gridDim, blockDim) (devStates, w, n);
	hipDeviceSynchronize();

	hipFree(devStates);
}
#endif
//============================================================================================

#ifdef __NVCC__
#endif 
