#include "Utils.h"

#ifdef __NVCC__
float* createFloatArrayGPU(int n)
{
	float* a = NULL;
	if (hipMalloc((void**)&a, n * sizeof(float)) != hipSuccess) {
		printf("Array GPU allocation error\n");
		return NULL;
	}
	else {
		hipMemset(a, 0, sizeof(float) * n);
		return a;
	}
}
#endif // __NVCC__